#include "hip/hip_runtime.h"
#include "preprocess_gpu.h"
#define BLOCKSIZE 16

#if 0
__device__ float bilinearInterpolateChannel(const unsigned char sharedImg[][BLOCKSIZE][3], int sharedWidth, int sharedHeight, float x, float y, int channel, const unsigned char borderValue = 114) {
    if (x < 0 || y < 0 || x >= sharedWidth - 1 || y >= sharedHeight - 1) {
        return static_cast<float>(borderValue);
    }

    int x1 = static_cast<int>(x);
    int y1 = static_cast<int>(y);
    int x2 = x1 + 1;
    int y2 = y1 + 1;

    float a = x - x1;
    float b = y - y1;

    // 修改索引计算，以适应共享内存
    int idx1 = (y1 * sharedWidth + x1) * CHANNELS + channel;
    int idx2 = (y1 * sharedWidth + x2) * CHANNELS + channel;
    int idx3 = (y2 * sharedWidth + x1) * CHANNELS + channel;
    int idx4 = (y2 * sharedWidth + x2) * CHANNELS + channel;

    // 从共享内存读取像素值
    float inter1 = (1 - a) * sharedImg[y1][x1][channel] + a * sharedImg[y1][x2][channel];
    float inter2 = (1 - a) * sharedImg[y2][x1][channel] + a * sharedImg[y2][x2][channel];

    return (1 - b) * inter1 + b * inter2;
}

__global__ void preprocess_kernel_shared(const unsigned char* input, float* output, const float* matrix, int width, int height, int w, int h, const float* mean, const float* std) {
    // 定义共享内存
    __shared__ unsigned char sharedInput[BLOCKSIZE][BLOCKSIZE][3];

    // 计算每个线程的全局索引
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // 加载数据到共享内存
    if (idx < width && idy < height) {
        for (int c = 0; c < 3; ++c) {
            int globalIndex = (idy * width + idx) * 3 + c;
            sharedInput[threadIdx.y][threadIdx.x][c] = input[globalIndex];
        }
    }

    __syncthreads();

    // 确保线程对应于输出图像内的像素
    if (idx >= w || idy >= h) return;

    // 计算映射后的坐标
    float x = matrix[0] * idx + matrix[1] * idy + matrix[2];
    float y = matrix[3] * idx + matrix[4] * idy + matrix[5];

    // 双线性插值和归一化
    int channelSize = w * h;
    int index = idy * w + idx;
    // output[index] = (bilinearInterpolateChannel(sharedInput, blockDim.x, blockDim.y, x, y, 2) / 255.0f - mean[0]) / std[0];
    // output[channelSize + index] = (bilinearInterpolateChannel(sharedInput, blockDim.x, blockDim.y, x, y, 1) / 255.0f - mean[1]) / std[1];
    // output[2 * channelSize + index] = (bilinearInterpolateChannel(sharedInput, blockDim.x, blockDim.y, x, y, 0) / 255.0f - mean[2]) / std[2];
    output[index] = (bilinearInterpolateChannel(sharedInput, width, height, x, y, 2, 3) / 255.0f - mean[0]) / std[0];
    output[channelSize + index] = (bilinearInterpolateChannel(sharedInput, width, height, x, y, 1, 3) / 255.0f - mean[1]) / std[1];
    output[2 * channelSize + index] = (bilinearInterpolateChannel(sharedInput, width, height, x, y, 0, 3) / 255.0f - mean[2]) / std[2];
}

#else
__device__ float bilinearInterpolateChannel(const unsigned char* img, int width, int height, float x, float y, int channel, int channels, const unsigned char borderValue = 114) {
    if (x < 0 || y < 0 || x >= width - 1 || y >= height - 1) {
        return static_cast<float>(borderValue);
    }

    int x1 = static_cast<int>(x);
    int y1 = static_cast<int>(y);
    int x2 = x1 + 1;
    int y2 = y1 + 1;

    float a = x - x1;
    float b = y - y1;

    int idx1 = (y1 * width + x1) * channels + channel;
    int idx2 = (y1 * width + x2) * channels + channel;
    int idx3 = (y2 * width + x1) * channels + channel;
    int idx4 = (y2 * width + x2) * channels + channel;

    float inter1 = (1 - a) * img[idx1] + a * img[idx2];
    float inter2 = (1 - a) * img[idx3] + a * img[idx4];

    return (1 - b) * inter1 + b * inter2;
}
__global__ void preprocess_kernel(const unsigned char* input, float* output, const float* matrix, int width, int height, int w, int h, const float* mean, const float* std) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= w && idy >= h) return ;
    float x = matrix[0] * idx + matrix[1] * idy + matrix[2];
    float y = matrix[3] * idx + matrix[4] * idy + matrix[5];

    int channelSize = w * h;
    int index = idy * w + idx;
    output[index] = (bilinearInterpolateChannel(input, width, height, x, y, 2, 3) / 255.0f - mean[0]) / std[0];
    output[channelSize + index] = (bilinearInterpolateChannel(input, width, height, x, y, 1, 3) / 255.0f - mean[1]) / std[1];
    output[2 * channelSize + index] = (bilinearInterpolateChannel(input, width, height, x, y, 0, 3) / 255.0f - mean[2]) / std[2];
}
#endif




int preprocess_gpu(const unsigned char* d_input, float* d_output, int original_width, int original_height, int target_width, int target_height, const float* d_matrix, const float* d_mean, const float* d_std) {
    // 设置 CUDA 网格和块的大小
    dim3 blockSize(BLOCKSIZE, BLOCKSIZE);
    dim3 gridSize((target_width + blockSize.x - 1) / blockSize.x, (target_height + blockSize.y - 1) / blockSize.y);
    // 调用 CUDA 内核
    preprocess_kernel<<<gridSize, blockSize>>>(d_input, d_output, d_matrix, original_width, original_height, target_width, target_height, d_mean, d_std);

    // 检查 CUDA 是否成功执行
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "Preprocess CUDA Error: " << hipGetErrorString(error) << std::endl;
        return -1;
    }
    // 同步 CUDA 设备以确保所有操作都已完成
    hipDeviceSynchronize();
    return 0;
}

